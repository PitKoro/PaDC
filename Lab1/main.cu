
#include <hip/hip_runtime.h>
#include <stdlib.h> 
#include <stdio.h> 

 
void Sin(float*, int, double); 
 
__global__ void SinKernel(float *a, float *b) { 
int idx = threadIdx.x + blockIdx.x * blockDim.x; 
b[idx] = sinf(a[idx]); 
} 
 
void Printer(float *a, int n){ 
 for (int i = 0; i < n; i++){ 
 printf("%f\n", a[i]); 
 } 
} 
 
void Assigner(float *a, int n){ 
 for (int i = 0; i < n; i++){ 
 a[i] = (float)i; 
 } 
} 
 
int main(){ 
 
int n = 1024 * 1024; 
int size = n * sizeof(double);
 
 
float *aDev = NULL, *bDev = NULL; 
float *a = NULL, *b = NULL; 
 
hipMalloc((void **) &aDev, size); 
hipMalloc((void **) &bDev, size); 
 
a = (float *) malloc(size); 
b = (float *) malloc(size); 
 
Assigner(a, n); 
 
dim3 threads = dim3(512, 1); 
dim3 blocks = dim3(n / threads.x, 1); 
 
hipMemcpy(aDev, a, size, hipMemcpyHostToDevice); 
hipMemcpy(bDev, b, size, hipMemcpyHostToDevice); 
 
SinKernel<<<blocks, threads>>> (aDev, bDev); 
 
hipMemcpy(b, bDev, size, hipMemcpyDeviceToHost); 
 
Printer(b, n); 
 
hipFree(aDev); 
hipFree(bDev); 
 
free(a); 
free(b); 
}


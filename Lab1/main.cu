
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

__global__ void sinX(double *x, double *result) {
    *result = sin(*x);
}

int main() {
    clock_t begin = clock();
    // переменные хоста
    double x, result;
    int size = sizeof(double);

    // копии для устройства
    double *d_x, *d_result;

    // выделяем память на устройстве
    hipMalloc((void **)&d_x, size);
    hipMalloc((void **)&d_result, size);

    // инициализируем переменную хоста
    x = 1;

    // копируем данные с хоста на устройство
    hipMemcpy(d_x, &x, size, hipMemcpyHostToDevice);
    hipMemcpy(d_result, &result, size, hipMemcpyHostToDevice);

    // вызов функции на хосте
    // но работать она будет на устройстве
    sinX<<<1,1>>>(d_x, d_result);

    //копируем данные с устройства на хост
    hipMemcpy(&result, d_result, size, hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_x); hipFree(d_result);
    clock_t end = clock();
    double time_spent = (double)(end - begin) / CLOCKS_PER_SEC;

    printf("GPU:\n%f\n", result);
    printf("Time: %f\n", time_spent);

    return 0;
}